#include "hip/hip_runtime.h"
// example.cu
//#include <pybind11/pybind11.h>
#include "/home/user_home/xiaomingxu/mambaforge/envs/ccdl/lib/python3.10/site-packages/pybind11/include/pybind11/pybind11.h"
#include <hip/hip_runtime.h>

__global__ void addKernel(int* a, int* b, int* c, int size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) {
        c[i] = a[i] + b[i];
    }
}

void add(int* a, int* b, int* c, int size) {
    int* dev_a;
    int* dev_b;
    int* dev_c;

    hipMalloc((void**)&dev_a, size * sizeof(int));
    hipMalloc((void**)&dev_b, size * sizeof(int));
    hipMalloc((void**)&dev_c, size * sizeof(int));

    hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;
    addKernel<<<blocksPerGrid, threadsPerBlock>>>(dev_a, dev_b, dev_c, size);

    hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
}


